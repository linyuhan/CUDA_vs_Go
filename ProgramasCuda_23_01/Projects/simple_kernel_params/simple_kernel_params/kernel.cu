#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "C:/Users/Lin/Documents/visual studio 2012/Projects/common/book.h"


#include <stdio.h>

__global__ void add( int a, int b, int *c ) {
    *c = a + b;
	printf("%d", a);
}

int main( void ) {
    int c;
    int *dev_c;
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, sizeof(int) ) );

    add<<<1,1>>>( 2, 7, dev_c );

    HANDLE_ERROR( hipMemcpy( &c, dev_c, sizeof(int),
                              hipMemcpyDeviceToHost ) );
    printf( "2 + 7 = %d\n", c );
    HANDLE_ERROR( hipFree( dev_c ) );

    return 0;
}
