#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <windows.h>

const int arraySize = 501;

void somaVetorCuda(const int *a, const int *b, int *c, unsigned int size);

void somaVetorLoop(int a[arraySize], int b[arraySize], int c[arraySize], int arraySize);

__global__ void somaKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	int inicio, final; 


	int a[arraySize];
	int b[arraySize];
	int c[arraySize];

	for (int i=0; i<arraySize;i++){
		a[i] = i;
		b[i] = i;
	}

    inicio = GetTickCount();
    // Add vectors in parallel.
    //somaVetorCuda(a, b, c, arraySize);
	somaVetorLoop(a, b, c, arraySize);

	final = GetTickCount();
    printf("\n\ntempo decorrido: %d\n", final-inicio);
	for (int i=0;i<arraySize;i++){
		printf("%d, ", c[i]);
	}

	

	

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    hipDeviceReset();

    return 0;
}

void somaVetorLoop(int a[arraySize], int b[arraySize], int c[arraySize], int arraySize){
	for (int i=0; i<arraySize;i++){
		c[i] = a[i] + b[i];
	}
}

// Fun��o que soma os vetores de modo paralelo
void somaVetorCuda(const int *a, const int *b, int *c, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
   
    // Allocate GPU buffers for three vectors (two input, one output)
    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    // Copy input vectors from host memory to GPU buffers.
    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Launch a kernel on the GPU with one thread for each element.
    somaKernel<<<1, size>>>(dev_c, dev_a, dev_b);
    
    // hipDeviceSynchronize waits for the kernel to finish
    //hipDeviceSynchronize();

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

}
